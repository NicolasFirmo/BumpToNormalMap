#include <hip/hip_runtime.h>
 // threadIdx
#include <hip/device_functions.h>
#include <cstdio>

static unsigned char* d_in;
static unsigned char* d_out;

static unsigned int h_Width;
static unsigned int h_Height;

static unsigned int h_BlockWidth;
static unsigned int h_BlockHeight;

#define THREAD_TOTAL_X_LEN 16
#define THREAD_AUX_X_LEN 2
#define THREAD_WORKING_X_LEN (THREAD_TOTAL_X_LEN - THREAD_AUX_X_LEN)

#define THREAD_TOTAL_Y_LEN 16
#define THREAD_AUX_Y_LEN 2
#define THREAD_WORKING_Y_LEN (THREAD_TOTAL_Y_LEN - THREAD_AUX_Y_LEN)

#define OFFSET(x,y) sIdx + y * THREAD_TOTAL_X_LEN + x

__global__ void Sobel(const unsigned char* in,unsigned char* out, const unsigned int width, const unsigned int height)
{
	extern __shared__ short s[];

	const unsigned int xPos = (blockIdx.x * THREAD_WORKING_X_LEN + threadIdx.x) - (THREAD_AUX_X_LEN / 2);
	const unsigned int yPos = (blockIdx.y * THREAD_WORKING_Y_LEN + threadIdx.y) - (THREAD_AUX_Y_LEN / 2);
	const unsigned int inPos = (xPos + yPos * width);
	const unsigned int sIdx = (threadIdx.x + threadIdx.y * THREAD_TOTAL_X_LEN);

	if (xPos < width && yPos < height)
		s[sIdx] = in[inPos];
	else
		s[sIdx] = 0;

	__syncthreads();

	if ((threadIdx.x - (THREAD_AUX_X_LEN / 2)) < THREAD_WORKING_X_LEN && (threadIdx.y - (THREAD_AUX_X_LEN / 2)) < THREAD_WORKING_Y_LEN)
	{
		const float sobelX = (
													-1 * s[OFFSET(-1,-1)] +0 * s[OFFSET(0,-1)] +1 * s[OFFSET(1,-1)]
													-2 * s[OFFSET(-1, 0)] +0 * s[OFFSET(0, 0)] +2 * s[OFFSET(1, 0)]
													-1 * s[OFFSET(-1, 1)] +0 * s[OFFSET(0, 1)] +1 * s[OFFSET(1, 1)]
												)/(0.08f * 255.0f);

		const float sobelY =	(
													+1 * s[OFFSET(-1,-1)] +2 * s[OFFSET(0,-1)] +1 * s[OFFSET(1,-1)]
													+0 * s[OFFSET(-1, 0)] +0 * s[OFFSET(0, 0)] +0 * s[OFFSET(1, 0)]
													-1 * s[OFFSET(-1, 1)] -2 * s[OFFSET(0, 1)] -1 * s[OFFSET(1, 1)]
												)/(0.08f * 255.0f);

		const float gradientLen = sqrt(sobelX*sobelX + sobelY*sobelY + 1.0f);

		const unsigned char xLen = (-sobelX/gradientLen) * 128.0f + 128.0f;
		const unsigned char yLen = (-sobelY/gradientLen) * 128.0f + 128.0f;
		const unsigned char zLen = 255.0f/gradientLen;

		out[inPos * 4 + 0] = xLen;
		out[inPos * 4 + 1] = yLen;
		out[inPos * 4 + 2] = zLen;
		out[inPos * 4 + 3] = 255;
	}
}

void InitBumpToNormalMap(const unsigned int width, const unsigned int height)
{
	h_Width = width;
	h_Height = height;

	h_BlockWidth = (h_Width / THREAD_WORKING_X_LEN);
	h_BlockHeight = (h_Height / THREAD_WORKING_Y_LEN);

	hipMalloc(&d_in, h_Width * h_Height * 1 * sizeof(unsigned char));
	hipMalloc(&d_out, h_Width * h_Height * 4 * sizeof(unsigned char));
}

int KernelBumpToNormalMap(unsigned char* h_in_img,unsigned char* h_out_img)
{
	hipError_t error = hipMemcpy(d_in, h_in_img, h_Width * h_Height * 1 * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (error != hipSuccess)
		return error;

	Sobel<<<dim3(h_BlockWidth, h_BlockHeight, 1), dim3(THREAD_TOTAL_X_LEN, THREAD_TOTAL_Y_LEN, 1), THREAD_TOTAL_X_LEN * THREAD_TOTAL_Y_LEN * sizeof(short)>>>(d_in, d_out, h_Width, h_Height);
	error = hipGetLastError();
	if (error != hipSuccess)
		return error;

	error = hipMemcpy(h_out_img, d_out, h_Width * h_Height * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (error != hipSuccess)
		return error;

	return hipSuccess;
}

void ShutdownBumpToNormalMap()
{
	hipFree(d_in);
	hipFree(d_out);
}

