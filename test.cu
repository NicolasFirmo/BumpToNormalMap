#include <hip/hip_runtime.h>
 // threadIdx
#include <hip/device_functions.h>
#include <cstdio>

static float* d_in;
static float* d_out;

static unsigned int h_Width;
static unsigned int h_Height;

static unsigned int h_BlockWidth;
static unsigned int h_BlockHeight;

#define THREAD_TOTAL_X_LEN 16
#define THREAD_AUX_X_LEN 2
#define THREAD_WORKING_X_LEN (THREAD_TOTAL_X_LEN - THREAD_AUX_X_LEN)

#define THREAD_TOTAL_Y_LEN 16
#define THREAD_AUX_Y_LEN 2
#define THREAD_WORKING_Y_LEN (THREAD_TOTAL_Y_LEN - THREAD_AUX_Y_LEN)

#define OFFSET(x,y) sIdx + y * THREAD_TOTAL_X_LEN + x

__global__ void Sobel(const float* in,float* out, const unsigned int width, const unsigned int height)
{
	extern __shared__ float s[];

	const unsigned int xPos = (blockIdx.x * THREAD_WORKING_X_LEN + threadIdx.x) - (THREAD_AUX_X_LEN / 2);
	const unsigned int yPos = (blockIdx.y * THREAD_WORKING_Y_LEN + threadIdx.y) - (THREAD_AUX_Y_LEN / 2);
	const unsigned int inPos = (xPos + yPos * width);
	const unsigned int sIdx = (threadIdx.x + threadIdx.y * THREAD_TOTAL_X_LEN);

	if (xPos < width && yPos < height)
		s[sIdx] = in[inPos];
	else
		s[sIdx] = 0;

	__syncthreads();

	if ((threadIdx.x - (THREAD_AUX_X_LEN / 2)) < THREAD_WORKING_X_LEN && (threadIdx.y - (THREAD_AUX_X_LEN / 2)) < THREAD_WORKING_Y_LEN)
	{
		const float sobelX = (
													-1 * s[OFFSET(-1,-1)] +0 * s[OFFSET(0,-1)] +1 * s[OFFSET(1,-1)]
													-2 * s[OFFSET(-1, 0)] +0 * s[OFFSET(0, 0)] +2 * s[OFFSET(1, 0)]
													-1 * s[OFFSET(-1, 1)] +0 * s[OFFSET(0, 1)] +1 * s[OFFSET(1, 1)]
												) * 0.25f;

		const float sobelY =	(
													+1 * s[OFFSET(-1,-1)] +2 * s[OFFSET(0,-1)] +1 * s[OFFSET(1,-1)]
													+0 * s[OFFSET(-1, 0)] +0 * s[OFFSET(0, 0)] +0 * s[OFFSET(1, 0)]
													-1 * s[OFFSET(-1, 1)] -2 * s[OFFSET(0, 1)] -1 * s[OFFSET(1, 1)]
												) * 0.25f;

		const float gradientLen = sqrt(sobelX*sobelX + sobelY*sobelY + 1.0f);

		const float xLen = (-sobelX/gradientLen) * 0.5f + 0.5f;
		const float yLen = (-sobelY/gradientLen) * 0.5f + 0.5f;
		const float zLen = 1.0f/gradientLen;

		out[inPos * 4 + 0] = xLen;
		out[inPos * 4 + 1] = yLen;
		out[inPos * 4 + 2] = zLen;
		out[inPos * 4 + 3] = 1.0f;
	}
}

void InitBumpToNormalMap(const unsigned int width, const unsigned int height)
{
	h_Width = width;
	h_Height = height;

	h_BlockWidth = (h_Width / THREAD_WORKING_X_LEN);
	h_BlockHeight = (h_Height / THREAD_WORKING_Y_LEN);

	hipMalloc(&d_in, h_Width * h_Height * 1 * sizeof(float));
	hipMalloc(&d_out, h_Width * h_Height * 4 * sizeof(float));
}

int KernelBumpToNormalMap(float* h_in_img,float* h_out_img)
{
	hipError_t error = hipMemcpy(d_in, h_in_img, h_Width * h_Height * 1 * sizeof(float), hipMemcpyHostToDevice);
	if (error != hipSuccess)
		return error;

	Sobel<<<dim3(h_BlockWidth, h_BlockHeight, 1), dim3(THREAD_TOTAL_X_LEN, THREAD_TOTAL_Y_LEN, 1), THREAD_TOTAL_X_LEN * THREAD_TOTAL_Y_LEN * sizeof(float)>>>(d_in, d_out, h_Width, h_Height);
	error = hipGetLastError();
	if (error != hipSuccess)
		return error;

	error = hipMemcpy(h_out_img, d_out, h_Width * h_Height * 4 * sizeof(float), hipMemcpyDeviceToHost);
	if (error != hipSuccess)
		return error;

	return hipSuccess;
}

void ShutdownBumpToNormalMap()
{
	hipFree(d_in);
	hipFree(d_out);
}

