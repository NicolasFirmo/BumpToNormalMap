#include <hip/hip_runtime.h>
 // threadIdx
#include <hip/device_functions.h>
#include <cstdio>

static unsigned char* d_in;
static unsigned char* d_out;

static unsigned int h_Width;
static unsigned int h_Height;

static unsigned int h_BlockWidth;
static unsigned int h_BlockHeight;

#define THREAD_TOTAL_X_LEN 16
#define THREAD_AUX_X_LEN 2
#define THREAD_WORKING_X_LEN (THREAD_TOTAL_X_LEN - THREAD_AUX_X_LEN)

#define THREAD_TOTAL_Y_LEN 16
#define THREAD_AUX_Y_LEN 2
#define THREAD_WORKING_Y_LEN (THREAD_TOTAL_Y_LEN - THREAD_AUX_Y_LEN)

#define OFFSET(x,y) sIdx + y * THREAD_TOTAL_X_LEN + x

__global__ void Sobel(const unsigned char* in,unsigned char* out, const unsigned int width, const unsigned int height)
{
	extern __shared__ short s[];

	const unsigned int xPos = (blockIdx.x * THREAD_WORKING_X_LEN + threadIdx.x) - (THREAD_AUX_X_LEN / 2);
	const unsigned int yPos = (blockIdx.y * THREAD_WORKING_Y_LEN + threadIdx.y) - (THREAD_AUX_Y_LEN / 2);
	const unsigned int inPos = (xPos + yPos * width);
	const unsigned int sIdx = (threadIdx.x + threadIdx.y * THREAD_TOTAL_X_LEN);
	unsigned int outIt = inPos * 4;

	if (xPos < width && yPos < height)
		s[sIdx] = in[inPos];
	else
		s[sIdx] = 0;

	__syncthreads();

	if ((threadIdx.x - (THREAD_AUX_X_LEN / 2)) < THREAD_WORKING_X_LEN && (threadIdx.y - (THREAD_AUX_X_LEN / 2)) < THREAD_WORKING_Y_LEN)
	{
		const short sobelX = (
													-1 * s[OFFSET(-1,-1)] +0 * s[OFFSET(0,-1)] +1 * s[OFFSET(1,-1)]
													-2 * s[OFFSET(-1, 0)] +0 * s[OFFSET(0, 0)] +2 * s[OFFSET(1, 0)]
													-1 * s[OFFSET(-1, 1)] +0 * s[OFFSET(0, 1)] +1 * s[OFFSET(1, 1)]
												)/4;

		const short sobelY =	(
													+1 * s[OFFSET(-1,-1)] +2 * s[OFFSET(0,-1)] +1 * s[OFFSET(1,-1)]
													+0 * s[OFFSET(-1, 0)] +0 * s[OFFSET(0, 0)] +0 * s[OFFSET(1, 0)]
													-1 * s[OFFSET(-1, 1)] -2 * s[OFFSET(0, 1)] -1 * s[OFFSET(1, 1)]
												)/4;

		const short gradientLen = sqrt(float(sobelX*sobelX + sobelY*sobelY + 255 * 255));

		const unsigned char xLen = -(sobelX * 128)/gradientLen + 128;
		const unsigned char yLen = -(sobelY * 128)/gradientLen + 128;
		const unsigned char zLen = (255 * 255)/gradientLen;

		out[outIt++] = xLen;
		out[outIt++] = yLen;
		out[outIt++] = zLen;
		out[outIt] = 255;
	}
}

void InitBumpToNormalMap(const unsigned int width, const unsigned int height)
{
	h_Width = width;
	h_Height = height;

	h_BlockWidth = (h_Width / THREAD_WORKING_X_LEN);
	h_BlockHeight = (h_Height / THREAD_WORKING_Y_LEN);

	hipMalloc(&d_in, h_Width * h_Height * 1 * sizeof(unsigned char));
	hipMalloc(&d_out, h_Width * h_Height * 4 * sizeof(unsigned char));
}

int KernelBumpToNormalMap(unsigned char* h_in_img,unsigned char* h_out_img)
{
	hipError_t error = hipMemcpy(d_in, h_in_img, h_Width * h_Height * 1 * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (error != hipSuccess)
		return error;

	Sobel<<<dim3(h_BlockWidth, h_BlockHeight, 1), dim3(THREAD_TOTAL_X_LEN, THREAD_TOTAL_Y_LEN, 1), THREAD_TOTAL_X_LEN * THREAD_TOTAL_Y_LEN * sizeof(short)>>>(d_in, d_out, h_Width, h_Height);
	error = hipGetLastError();
	if (error != hipSuccess)
		return error;

	error = hipMemcpy(h_out_img, d_out, h_Width * h_Height * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (error != hipSuccess)
		return error;

	return hipSuccess;
}

void ShutdownBumpToNormalMap()
{
	hipFree(d_in);
	hipFree(d_out);
}

